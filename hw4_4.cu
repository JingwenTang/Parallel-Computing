/*
========================================================================================
Name: cuda_demo.cu
Author: Mingran Peng
Class: EECS 587, Fall 2020
Description : Demo program for HW4
P.S. Fell free to use or modify this code for future terms of EECS 587 or other courses
Add you name if you modify it and preserve all author names
========================================================================================
*/

#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <cstdio>
#include <cmath>
#include <numeric>
#include <iostream>
#include <cstring>
#include <sstream>
#include <string>

using namespace std;

hipError_t addWithCuda(double *c, unsigned int size, int n, double *Along, int t);

__global__ void addKernel(double *c, int size, int n, double *dev_Along, int t)
{
    double arr[4];
    double secondSmallest=0;
    double temp;
    double update;
    // calculate index here
    int localro = threadIdx.x;
    int localco = threadIdx.y;
    int globalro = blockIdx.x * blockDim.x + threadIdx.x;
    int globalco = blockIdx.y * blockDim.y + threadIdx.y;
    int globalind = globalro * n + globalco;
    // Synchronize all threads in the block to make sure copy is done
    __syncthreads();
    if ((globalro>0) && (globalco>0) && (globalro<(n-1)) && (globalco<(n-1))){
        int il = globalro+1;
        int is = globalro-1;
        int jl = globalco+1;
        int js = globalco-1;
        arr[0] = dev_Along[il*n+jl];
        arr[1] = dev_Along[il*n+js];
        arr[2] = dev_Along[is*n+jl];
        arr[3] = dev_Along[is*n+js];
        for(int i=0;i<4;i++)
            {
                for(int j=i+1;j<4;j++)
                {
                    if(arr[i]>arr[j])
                    {
                        temp  =arr[i];
                        arr[i]=arr[j];
                        arr[j]=temp;
                    }
                }
            }
        secondSmallest = arr[1];
        update = dev_Along[globalind] + secondSmallest;
    }
    else if ((globalro==0)||(globalco==0)||(globalro==(n-1))||(globalco==(n-1))){
        update = dev_Along[globalind];
    }
    else{}
    if((globalro<n)&&(globalco<n))
        c[globalind] = update;
}

int main(int argc, char* argv[])
{
    int n;
    int t;
    n = atoi(argv[1]);
    t = atoi(argv[2]);

    int size = n*n;//number of elements
    double *c; // returned array
    //initiallize
    double *Along;
    Along = new double [size];
    c = new double [size];
    for (int i=0;i<size;i++){
        int ro = floor(i/n);
        int co = floor(i%n);
        Along[i] = pow((1+cos(2*ro)+sin(co)),2);
    }


    hipError_t cudaStatus = addWithCuda(c, size, n, Along, t);
    if (cudaStatus != hipSuccess) {
        cout<<"addWithCuda failed!"<<endl;
        return -1;
    }
    
    //here we get the c array then we can do the sum and check the certain element
    // examine
    double initial_sum = 0;
    double sumc = accumulate(c, c+size, initial_sum);
    cout<<"Sum: "<<sumc<<endl;
    cout<<"A(37,47): "<<c[37*n+47]<<endl;
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double *c, unsigned int size, int n, double *Along, int t)
{
    double *dev_Along = 0;
    //dev_Along = new double[size];
    double *dev_c = 0;

    dev_c = new double[size];
    for (int i =0;i<size;i++){
        dev_c[i]=0;
    }
    hipError_t cudaStatus;
    hipEvent_t start, stop;
    float gpu_time = 0.0f;
    dim3 gridSize(ceil(n/32)+1,ceil(n/32)+1,1);
    dim3 blockSize(32,32,1);


    // Choose which GPU to run on, 0 if you have only one GPU
    // on-chip GPU does not count
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        cout<<"hipSetDevice failed!"<<endl;
        goto Error;
    }
    // Malloc memory on GPU
    
    
    cudaStatus = hipMalloc((void**)&dev_Along, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        cout<<"hipMalloc failed!"<<endl;
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        cout<<"hipMalloc failed!"<<endl;
        goto Error;
    }
    
    // Copy memory from Host to Device
    cudaStatus = hipMemcpy(dev_Along, Along, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        cout<<"hipMemcpy failed!"<<endl;
        goto Error;
    }
    // Set up timing
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // Launch a kernel on the GPU with one thread for each element
    //cout<<"allocating "<<gridSize<<" blocks, "<<blockSize<<" threads per block"<<endl;
    for (int tt=0;tt<t;tt++){
        addKernel<<<gridSize, blockSize>>>(dev_c, size, n,dev_Along,t);

        if (cudaStatus != hipSuccess) {
            cout<<"hipMemcpy failed!"<<endl;
            goto Error;
        }
        cudaStatus = hipMemcpy(dev_Along, dev_c, size * sizeof(double), hipMemcpyDeviceToDevice);
    }
    
    
    
      // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        cout<<"addKernel failed: "<<hipGetErrorString(cudaStatus)<<endl;
        goto Error;
    }
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        cout<<"hipDeviceSynchronize failed: "<<hipGetErrorString(cudaStatus)<<endl;
        goto Error;
    }
    // Close timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    cout<<"Time spent: "<<gpu_time<<"ms"<<endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy memory from devide to host
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        cout<<"hipMemcpy failed!"<<endl;
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_Along);


    return cudaStatus;
}
